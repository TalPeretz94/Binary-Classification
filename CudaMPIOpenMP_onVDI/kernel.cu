#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "point.h"
#include <stdio.h>

#define THREAD_AMOUNT 1024


Point* initCudaMalloc(const int N, Point* point, const int k);
int CalcNmissPoints(const int N, Point* point, float *w, const int k);

__device__ int getThreadId()
{
	return blockIdx.x *blockDim.x + threadIdx.x; //returning theard num (from internet)
}

__global__ void backprobagation(float *w, const float *point, const unsigned int size, const float alpha, const int sign)// update W in case of point not properly classified
{
	int i = getThreadId();
	if (i < size)
		w[i] = w[i] + alpha*sign*point[i];
}

__global__ void calcMiss(Point* point, float *w, const int k, const int N, int* nMiss)//each thread calc f(Xi) check if the signs are equals in case the sign not equals
																					  //the point not properly classified so we increase the amount of Nmiss.	
{
	int i = getThreadId();
	

	if (i < N) {
		float result = 0;
		for (int j = 0; j < k + 1; j++) {
			result = result + (point[i].value[j] * w[j]);
			
		}
		if (result*point[i].pointClass < 0) {
			//point[i].cuda = 1;
			atomicAdd(nMiss, 1);
			

		}
//		else
			//point[i].cuda = result; //

	}
}




Point* initCudaMalloc(const int N, Point* point, const int k) {
	Point *cuda_point;
	hipMalloc(&cuda_point, N * sizeof(Point));
	hipMemcpy(cuda_point, point, N * sizeof(Point), hipMemcpyHostToDevice);
	int i = 0;
	for (i = 0; i < N; i++) {
		float* arr;
		hipMalloc(&arr, (k + 1) * sizeof(float));
		hipMemcpy(&(cuda_point[i].value), &arr, sizeof(float*), hipMemcpyHostToDevice);
		hipMemcpy(arr, point[i].value, (k + 1) * sizeof(float), hipMemcpyHostToDevice);
	}
	return cuda_point;
}

int CalcNmissPoints(const int N, Point* point, float *w, const int k)
{
	float *cuda_w;
	
	int nMiss = 0;
	int *nMissCuda;
	hipMalloc(&cuda_w, (k+1) * sizeof(float));
	
	hipMalloc(&nMissCuda, sizeof(int));

	hipMemcpy(cuda_w, w, (k + 1) * sizeof(float), hipMemcpyHostToDevice);
	
	

	hipMemcpy(nMissCuda, &nMiss, sizeof(int), hipMemcpyHostToDevice);

	calcMiss << <N / THREAD_AMOUNT + 1, THREAD_AMOUNT >> > (point, cuda_w, k, N, nMissCuda);

	hipMemcpy(&nMiss, nMissCuda, sizeof(int), hipMemcpyDeviceToHost);
	/*hipMemcpy(point, cuda_point, N * sizeof(Point), hipMemcpyDeviceToHost);
	for (int i = 0; i < k; i++)
		hipMemcpy(point[i].po, cuda_point[i].po, N * sizeof(Point), hipMemcpyDeviceToHost);*/

	hipFree(cuda_w);
	
	hipFree(nMissCuda);

	return nMiss;
}
